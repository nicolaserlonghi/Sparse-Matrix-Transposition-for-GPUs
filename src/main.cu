#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <utilities.h>
#include <serial.h>
#include <nvidia.h>
#include <scanTrans.h>

int checkResults(int m, int *arrayA, int *arrayB);
int checkResults(int m, double *arrayA, double *arrayB);

int main(int argc, char **argv) {

    // Recupero titolo file input
    char    *filename = detectFile(argc, argv[1]);    
    int     m;
    int     n;
    int     nnz;    
    int     *csrRowPtr;
    int     *csrColIdx;
    double  *csrVal;

    // Recupero della matrice
    readMatrix(
                filename,
                m,
                n,
                nnz,
                csrRowPtr,
                csrColIdx,
                csrVal
            );

    int     *serialCscRowIdx  = (int *)malloc(nnz * sizeof(int));
    int     *serialCscColPtr  = (int *)malloc((n + 1) * sizeof(int));
    double  *serialCscVal     = (double *)malloc(nnz * sizeof(double));
    float   serialTime;
    
    // Esecuzione dell'algoritmo di trasposizione seriale
    serialTime = performTransposition(
                                    serial,
                                    m,
                                    n,
                                    nnz,
                                    csrRowPtr,
                                    csrColIdx,
                                    csrVal,
                                    serialCscColPtr,
                                    serialCscRowIdx,
                                    serialCscVal
                                );

    hipDeviceReset();
    std::cout << std::endl;

    // int     *nvidiaCscRowIdx  = (int *)malloc(nnz * sizeof(int));
    // int     *nvidiaCscColPtr  = (int *)malloc((n + 1) * sizeof(int));
    // double  *nvidiaCscVal     = (double *)malloc(nnz * sizeof(double));
    // float   nvidiaTime;

    // Esecuzione dell'algoritmo di trasposizione versione Nvidia ALGO1
    // nvidiaTime = performTransposition(
    //                                 nvidia,
    //                                 m,
    //                                 n,
    //                                 nnz,
    //                                 csrRowPtr,
    //                                 csrColIdx,
    //                                 csrVal,
    //                                 nvidiaCscColPtr,
    //                                 nvidiaCscRowIdx,
    //                                 nvidiaCscVal
    //                             );
    // if(nvidiaTime == -1) {
    //     std::cout << "GPU Sparse Matrix Transpostion ALGO1: memory is too low" << std::endl;
    //     std::cout << "ALGO1 speedup: -" << std::endl;
    // } else {
    //     std::cout << std::setprecision(1) << "ALGO1 speedup: " << serialTime / nvidiaTime << "x" << std::endl;
    // }    

    // hipDeviceReset();

    // free(nvidiaCscRowIdx);
    // free(nvidiaCscColPtr);
    // free(nvidiaCscVal);

    // std::cout << std::endl;

    // int     *nvidia2CscRowIdx  = (int *)malloc(nnz * sizeof(int));
    // int     *nvidia2CscColPtr  = (int *)malloc((n + 1) * sizeof(int));
    // double  *nvidia2CscVal     = (double *)malloc(nnz * sizeof(double));
    // float   nvidia2Time;

    // Esecuzione dell'algoritmo di trasposizione versione Nvidia ALGO2
    // nvidia2Time = performTransposition(
    //                             nvidia2,
    //                             m,
    //                             n,
    //                             nnz,
    //                             csrRowPtr,
    //                             csrColIdx,
    //                             csrVal,
    //                             nvidia2CscColPtr,
    //                             nvidia2CscRowIdx,
    //                             nvidia2CscVal
    //                         ); 

    // if(nvidia2Time == -1) {
    //     std::cout << "GPU Sparse Matrix Transpostion ALGO2: memory is too low" << std::endl;
    //     std::cout << "ALGO2 speedup: -" << std::endl;
    // } 
    // else {
    //     std::cout << std::setprecision(1) << "ALGO2 speedup: " << serialTime / nvidia2Time << "x" << std::endl;
    // }

    

    // hipDeviceReset();

    // free(nvidia2CscColPtr);
    // free(nvidia2CscRowIdx);
    // free(nvidia2CscVal); 

    // std::cout << std::endl;


    int     *scanTransCscRowIdx  = (int *)malloc(nnz * sizeof(int));
    int     *scanTransCscColPtr  = (int *)malloc((n + 1) * sizeof(int));
    double  *scanTransCscVal     = (double *)malloc(nnz * sizeof(double));
    float   scanTransTime;

    // Esecuzione dell'algoritmo di trasposizione versione articolo scanTrans
    scanTransTime = performTransposition(
                                        scanTrans,
                                        m,
                                        n,
                                        nnz,
                                        csrRowPtr,
                                        csrColIdx,
                                        csrVal,
                                        scanTransCscColPtr,
                                        scanTransCscRowIdx,
                                        scanTransCscVal
    ); 

    if(scanTransTime == -1) {
        std::cout << "GPU Sparse Matrix Transpostion ScanTrans: memory is too low" << std::endl;
        std::cout << "ScanTrans wrong: -" << std::endl;
        std::cout << "ScanTrans speedup: -" << std::endl;
    } 

    if(scanTransTime == -2) {
        std::cout << "GPU Sparse Matrix Transpostion ScanTrans: max blocks num reached" << std::endl;
        std::cout << "ScanTrans wrong: -" << std::endl;
        std::cout << "ScanTrans speedup: -" << std::endl;
    } 

    if(scanTransTime == -3) {
        std::cout << "GPU Sparse Matrix Transpostion ScanTrans: max threads num reached" << std::endl;
        std::cout << "ScanTrans wrong: -" << std::endl;
        std::cout << "ScanTrans speedup: -" << std::endl;
    } 

    if (scanTransTime != -1) {
        std::cout << std::setprecision(1) << "ScanTrans speedup: " << serialTime / scanTransTime << "x" << std::endl;
        std::cout << "check cscColPtr ScanTrans ";
        // scanTransTime = checkResults(n + 1, serialCscColPtr, scanTransCscColPtr);            
    }

    if (scanTransTime != -1) {
        std::cout << "\ncheck cscRowIdx ScanTrans ";
        // scanTransTime = checkResults(nnz, serialCscRowIdx, scanTransCscRowIdx);
    }

    if (scanTransTime != -1) {
        std::cout << "\ncheck cscVal ScanTrans ";
        // scanTransTime = checkResults(nnz, serialCscVal, scanTransCscVal);
    }

    if (scanTransTime != -1) {
        std::cout << "wrong: 0" << std::endl;
    }    

    hipDeviceReset();

    free(scanTransCscRowIdx);
    free(scanTransCscColPtr);
    free(scanTransCscVal);

    std::cout << std::endl;

    free(csrRowPtr); 
    free(csrColIdx); 
    free(csrVal);

    // free(serialCscRowIdx);
    // free(serialCscColPtr);
    // free(serialCscVal);   
}

int checkResults(int m, int *arrayA, int *arrayB) {
    for (int i = 0; i < m; i++) {
        if (arrayA[i] != arrayB[i]) {
            std::cout << "wrong: 1 \n";
                    //   << "\nhost:   " << arrayA[i]
                    //   << "\ndevice: " << arrayB[i] << "\n\n";
            hipDeviceReset();
            return -1;
        }
    }
    return 0;
    // std::cout << "\n<> Correct\n";
}

int checkResults(int m, double *arrayA, double *arrayB) {
    for (int i = 0; i < m; i++) {
        if (arrayA[i] != arrayB[i]) {
            std::cout << "wrong: 1 \n";
                    //   << "\nhost:   " << arrayA[i]
                    //   << "\ndevice: " << arrayB[i] << "\n\n";
            hipDeviceReset();
            return -1;
        }
    }
    return 0;
    // std::cout << "\n<> Correct\n";
}