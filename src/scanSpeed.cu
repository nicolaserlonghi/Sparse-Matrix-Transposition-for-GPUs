#include "hip/hip_runtime.h"
#include <scanSpeed.h>
#include <utilities.h>
#include <Timer.cuh>
#include "CheckError.cuh"

using namespace timer;

const int NUM_THREADS = 128;

// A helper macro to simplify handling cuda error checking
#define CUDA_ERROR( err, msg ) { \
    if (err != hipSuccess) {\
        printf( "%s: %s in %s at line %d\n", msg, hipGetErrorString( err ), __FILE__, __LINE__);\
        exit( EXIT_FAILURE );\
    }\
}

__global__
void transpostionSpeedKernel(
    int     m,
    int     n,
    int     nnz,
    int     *csrRowPtr,
    int     *csrColIdx,
    int     *cscColPtr,
    int     *csrRowIdx,
    int     *intra,
    int     *inter,
    int     inter_dim
) {

    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = blockDim.x * gridDim.x;
    int start;
    

    // prendo l'equivalente di biggest della versione seriale
    int  bestNumThread = n > m ? n : m;
    // se biggest è maggiore del numero di thread che ho allocato vuol dire che sarei uscita
    // quindi prendo nthreads (altrimenti detto come "prendo il più piccolo tra bestNumThread e nthreads")
    bestNumThread = inter_dim;
    if (global_id == 0) {
        printf("%d\n", bestNumThread);
    }
    
    int len;

    // if(global_id < bestNumThread) {
    //     len = nnz / bestNumThread;        
    //     // partizioniamo il numero di nnz tra i thread
    //     if ( global_id < nnz % bestNumThread) {                
    //         len++;
    //         start = len * global_id;                 
    //     }
    //     else {
    //         start = len * global_id + (nnz % bestNumThread);    
    //     }
        
    //     if (global_id < m) {
    //         for(int j = csrRowPtr[global_id]; j < csrRowPtr[global_id + 1]; j++) {
    //             csrRowIdx[j] = global_id;
    //         }
    //     }       
        
    //     for(int i = 0; i < len; i++) {
    //         int index = csrColIdx[start + i];
    //         int brutto = (global_id + 1) * n + index;
    //         intra[start + i] = inter[brutto];
    //         inter[brutto]++;
    //     }
    //     // global_id += nthreads;

    

    //     // for(int i = 0; i < len; i++) {
    //     //     int index = csrColIdx[start + i];
    //     //     long long int a = global_id;
    //     //     long long int b = n;
    //     //     long long int c = index;
    //     //     long long int brutto = (a + 1) * b + c;
    //     //     if(brutto > global_id || brutto < 0) {
    //     //         int *ptr = &inter[brutto];
    //     //         // if (x%2048)  ptr = &inter[((global_id + 1) * x + index) + 1];
    //     //         brutto = (global_id + 1) * x + index;
    //     //         intra[start + i] = ptr[brutto];
    //     //         ptr[brutto]++;
    //     //     } else {
    //     //         intra[start + i] = inter[brutto];
    //     //         inter[brutto]++;
    //     //     }
    //     // }
    //     // global_id += nthreads;
    // }




    /* int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = blockDim.x * gridDim.x;
    int start;
    

    int  bestNumThread = n > m ? n : m;
    int len;

    if (global_id == 0) {
        printf("%ld\n", n * (m - 2));
        printf("%ld\n", n * (m - 1));
        printf("%ld\n", n * (m));
        printf("%d\n", n * (m +1));
    }

    while(global_id < bestNumThread) {
        len = nnz / bestNumThread;
        // printf("best: %d, global_id: %d, len: %d\n", bestNumThread, global_id, len);        
        // partizioniamo il numero di nnz tra i thread
        if ( global_id < nnz % bestNumThread) {                
            len++;
            start = len * global_id;                 
        }
        else {
            start = len * global_id + (nnz % bestNumThread);    
        }
        
        if (global_id < m) {
            for(int j = csrRowPtr[global_id]; j < csrRowPtr[global_id + 1]; j++) {
                csrRowIdx[j] = global_id;
            }
        }        
     

        for(int i = 0; i < len; i++) {
            long index = csrColIdx[start + i];
            // if ( ((global_id + 1) * n + index ) < ((global_id + 1) * n + n) ) {
            
                // printf("%d - %d - %d - %d - %d\n", global_id, len, csrColIdx[start + i], start, (global_id + 1) * n);
                // int index = 0;
                // printf("%d =  %d * %d + %d --> len: %d\n", (global_id + 1) * n + index, global_id, n, index, len);
                long a = (global_id + 1) * n + index;
                // printf("global_id: %d, -> (%d + 1) * %d + %d =  a: %ld --> len: %d, start: %d\n", global_id, global_id, n, index, a, len, start);
                // intra[start + i] = *(inter + a);
                // *(inter + a) += 1;
            // }
        } 
        // printf("pref: %d, after: %d \n", global_id, global_id + nthreads);
        global_id += nthreads;
    } */
}

__global__
void transpostionSpeedKernel1(
    int     m,
    int     n,
    int     nnz,
    int     *csrColIdx,
    int     *cscColPtr,
    int     *csrRowIdx,
    int     *intra,
    int     *inter
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = blockDim.x * gridDim.x;

    if (global_id == 0) {
        printf("Ciao");
    }
    
    int  bestNumThread = n > m ? n : m;
    while(global_id < bestNumThread) {
        if (global_id < n) {
            for(int j = 1; j < (bestNumThread + 1); j++) {
                //inter[global_id + (n * j)] += inter[global_id + (n * (j-1))];
            }
        }
        global_id += nthreads;
    }
}



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




// Takes the output array and for each block i, adds value i from INCR array to every element
__global__
void uniformAdd(int *outputArray, int numElements, int *INCR){
	int index = threadIdx.x + (2 * NUM_THREADS) * blockIdx.x;

	int valueToAdd = INCR[blockIdx.x];

	// Each thread sums two elements
	if (index < numElements){
		outputArray[index] += valueToAdd;
	}
	if (index + NUM_THREADS < numElements){
		outputArray[index + NUM_THREADS] += valueToAdd;
	}
}

// Block prescan that works on any array length on NUM_THREADS * 2 length blocks
__global__
void blockPrescan(int *g_idata, int *g_odata, int n, int *SUM)
{
	__shared__ int temp[NUM_THREADS << 1]; // allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;
	int blockOffset = NUM_THREADS * blockIdx.x * 2;

//	 Copy the correct elements form the global array
	if (blockOffset + (thid * 2) < n){
        temp[thid * 2] = g_idata[blockOffset + (thid * 2)];
	}
	if (blockOffset + (thid * 2) + 1 < n){
        temp[(thid * 2)+1] = g_idata[blockOffset + (thid * 2)+1];
	}

//	 Build sum in place up the tree
	for (int d = NUM_THREADS; d > 0; d >>= 1){
		__syncthreads();

		if (thid < d){
			int ai = offset*((thid * 2)+1)-1;
			int bi = offset*((thid * 2)+2)-1;
			temp[bi] += temp[ai];
		}
		offset <<= 1;
	}

	if (thid == 0) {
		if(SUM != NULL){
			// If doing a FULL scan, save the last value in the SUMS array for later processing
			SUM[blockIdx.x] = temp[(NUM_THREADS << 1) - 1];
		}
		temp[(NUM_THREADS << 1) - 1] = 0; // clear the last element
	}

//	 Traverse down tree & build scan
	for (int d = 1; d < NUM_THREADS << 1; d <<= 1){
		offset >>= 1;
		__syncthreads();

		if (thid < d){
			int ai = offset*((thid * 2)+1)-1;
			int bi = offset*((thid * 2)+2)-1;

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

//	 Copy the new array back to global array
	__syncthreads();
	if (blockOffset + (thid * 2) < n){
        g_odata[blockOffset + (thid * 2)] = temp[(thid * 2)]; // write results to device memory
	}
	if (blockOffset + (thid * 2) + 1 < n){
        g_odata[blockOffset + ((thid * 2)+1)] = temp[(thid * 2)+1];
	}
}


void fullPrescan(int *d_input, int *d_cscColPtr, int numElements) {

    
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	size_t size = numElements * sizeof(int);

	// The number of blocks it would take to process the array at each level
	int blocksPerGridL1 = 1 + (numElements - 1) / (NUM_THREADS * 2);
	int blocksPerGridL2 = 1 + blocksPerGridL1 / (NUM_THREADS * 2);
    int blocksPerGridL3 = 1 + blocksPerGridL2 / (NUM_THREADS * 2);


	// int *d_input = NULL;
	// err = hipMalloc((void **) &d_input, size);
    // CUDA_ERROR(err, "Failed to allocate device array x");    

	// int *d_cscColPtr = NULL;
	// err = hipMalloc((void**) &d_cscColPtr, size);
    // CUDA_ERROR(err, "Failed to allocate device array y");

	// Only define in here and actually allocate memory to these arrays if needed
	int *d_SUMS_LEVEL1 = NULL;
	int *d_INCR_LEVEL1 = NULL;
	int *d_SUMS_LEVEL2 = NULL;
	int *d_INCR_LEVEL2 = NULL;

	// err = hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    // CUDA_ERROR(err, "Failed to copy array x from host to device");


    //-----------------Pick the correct level and execute the kernels----------
    
    Timer<DEVICE> TM_kernelNew;
    TM_kernelNew.start();

	// The correct level is going to be where the SUMS array can be prescanned with only one block
	if(blocksPerGridL1 == 1){
	    blockPrescan<<<blocksPerGridL1, NUM_THREADS>>>(d_input, d_cscColPtr, numElements, NULL);
        hipDeviceSynchronize();
	} else if (blocksPerGridL2 == 1) {
        
		// SUMS and INCR arrays need to be allocated to store intermediate values
        err = hipMalloc((void**) &d_SUMS_LEVEL1, blocksPerGridL1 * sizeof(int));
		CUDA_ERROR(err, "Failed to allocate device vector d_SUMS_LEVEL1");

		err = hipMalloc((void**) &d_INCR_LEVEL1, blocksPerGridL1 * sizeof(int));
        CUDA_ERROR(err, "Failed to allocate device vector d_INCR_LEVEL1");
        
		blockPrescan<<<blocksPerGridL1, NUM_THREADS>>>(d_input, d_cscColPtr, numElements, d_SUMS_LEVEL1);

		// Run a second prescan on the SUMS array
		blockPrescan<<<blocksPerGridL2, NUM_THREADS>>>(d_SUMS_LEVEL1, d_INCR_LEVEL1, blocksPerGridL1, NULL);

		// Add the values of INCR array to the corresponding blocks of the d_cscColPtr array
		uniformAdd<<<blocksPerGridL1, NUM_THREADS>>>(d_cscColPtr, numElements, d_INCR_LEVEL1);

		hipDeviceSynchronize();

	} else if (blocksPerGridL3 == 1) {
		// SUMS and INCR arrays need to be allocated to store intermediate values
		err = hipMalloc((void**) &d_SUMS_LEVEL1, blocksPerGridL1 * sizeof(int));
		CUDA_ERROR(err, "Failed to allocate device vector d_SUMS_LEVEL1");

		err = hipMalloc((void**) &d_SUMS_LEVEL2, (NUM_THREADS * 2) * sizeof(int));
		CUDA_ERROR(err, "Failed to allocate device vector d_SUMS_LEVEL2");

		err = hipMalloc((void**) &d_INCR_LEVEL1, blocksPerGridL1 * sizeof(int));
		CUDA_ERROR(err, "Failed to allocate device vector d_INCR");

		err = hipMalloc((void**) &d_INCR_LEVEL2, (NUM_THREADS * 2)* sizeof(int));
		CUDA_ERROR(err, "Failed to allocate device vector d_INCR");

		blockPrescan<<<blocksPerGridL1, NUM_THREADS>>>(d_input, d_cscColPtr, numElements, d_SUMS_LEVEL1);

		blockPrescan<<<blocksPerGridL2, NUM_THREADS>>>(d_SUMS_LEVEL1, d_INCR_LEVEL1, blocksPerGridL1, d_SUMS_LEVEL2);

		blockPrescan<<<blocksPerGridL3, NUM_THREADS>>>(d_SUMS_LEVEL2, d_INCR_LEVEL2, blocksPerGridL2, NULL);

		uniformAdd<<<blocksPerGridL2, NUM_THREADS>>>(d_INCR_LEVEL1, blocksPerGridL1, d_INCR_LEVEL2);

		uniformAdd<<<blocksPerGridL1, NUM_THREADS>>>(d_cscColPtr, numElements, d_INCR_LEVEL1);

		hipDeviceSynchronize();
	}else {
		printf("The array of length = %d is to large for a level 3 FULL prescan\n", numElements);
    }

    TM_kernelNew.stop();
    TM_kernelNew.print("Kernel_new: ");
    
    //---------------------------Timing and verification-----------------------

    err = hipGetLastError();
    CUDA_ERROR(err, "Failed to launch fullPrescan");

    // err = hipMemcpy(cscColPtr, d_cscColPtr, size, hipMemcpyDeviceToHost);
    // CUDA_ERROR(err, "Failed to copy array y from device to host");


	//-------------------------------Cleanup-----------------------------------
	// Free device memory
	// err = hipFree(d_input);
	// CUDA_ERROR(err, "Failed to free device array x");
	// err = hipFree(d_cscColPtr);
    // CUDA_ERROR(err, "Failed to free device array y");

	// Only need to free these arrays if they were allocated
	if(blocksPerGridL2 == 1 || blocksPerGridL3 == 1){
		err = hipFree(d_SUMS_LEVEL1);
		CUDA_ERROR(err, "Failed to free device array d_SUMS_LEVEL1");
		err = hipFree(d_INCR_LEVEL1);
		CUDA_ERROR(err, "Failed to free device array d_INCR_LEVEL1");
	}
	if(blocksPerGridL3 == 1){
		err = hipFree(d_SUMS_LEVEL2);
		CUDA_ERROR(err, "Failed to free device array d_SUMS_LEVEL2");
		err = hipFree(d_INCR_LEVEL2);
		CUDA_ERROR(err, "Failed to free device array d_INCR_LEVEL2");
    }

	// Reset the device
	//err = hipDeviceReset();
    //CUDA_ERROR(err, "Failed to reset the device");
}


int manageMemoryForScan(int numElements){
    // The number of blocks it would take to process the array at each level
	int blocksPerGridL1 = 1 + (numElements - 1) / (NUM_THREADS * 2);
	int blocksPerGridL2 = 1 + blocksPerGridL1 / (NUM_THREADS * 2);
    int blocksPerGridL3 = 1 + blocksPerGridL2 / (NUM_THREADS * 2);

    double nvidiaFreeMemory = getSizeOfNvidiaFreeMemory();
    int clean = 1;
    if (blocksPerGridL1 != 1 && blocksPerGridL2 == 1) {
        double occupancy = (blocksPerGridL1 * sizeof(int)) * 2;
        if((nvidiaFreeMemory - occupancy) < 0)
            clean = 0;
    } else if(blocksPerGridL1 != 1 && blocksPerGridL3 == 1) {
        double occupancy = ((blocksPerGridL1 + (NUM_THREADS * 2)) * sizeof(int)) * sizeof(int);
        if((nvidiaFreeMemory - occupancy) < 0)
            clean = 0;
    }
    return clean;
}



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////








__global__ 
void transpostionSpeedKernel3(
    int     m,
    int     n,
    int     nnz,
    int     *csrColIdx,
    double  *csrVal,
    int     *cscColPtr,
    int     *cscRowIdx,
    double  *cscVal,
    int     *csrRowIdx,
    int     *intra,
    int     *inter
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;    
    int nthreads = blockDim.x * gridDim.x;
    int start;    
    int  bestNumThread = n > m ? n : m;
    int len;
  
    while(global_id < bestNumThread) {
        len = nnz / bestNumThread;
        if ( global_id < nnz % bestNumThread) {
            len ++;
            start = len * global_id;    
        }
        else {
            start = len * global_id + (nnz % bestNumThread);
        }

        int loc;
        for(int i = 0; i < len; i++) {
            int row_offset = csrColIdx[start + i];
            int index = inter[global_id * n + row_offset];
            loc = cscColPtr[row_offset] + index + intra[start + i];
            cscRowIdx[loc] = csrRowIdx[start + i];
            cscVal[loc] = csrVal[start + i];        
        }

        global_id += nthreads;
    }
}


float scanSpeed(
    int     m,
    int     n,
    int     nnz,
    int     *csrRowPtr,
    int     *csrColIdx,
    double  *csrVal,
    int     *cscColPtr,
    int     *cscRowIdx,
    double  *cscVal
) {
    Timer<DEVICE> TM_device;

    int biggest = m > n ? m : n;
    int inter_dim = biggest;

    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    int maxThreads = props.multiProcessorCount * props.maxThreadsPerMultiProcessor;
    
    if (biggest > maxThreads) {
        inter_dim = 1000;
    }
    std::cout << biggest << std::endl;
    

    // int nthreads = biggest / NUM_THREADS;
    // if(biggest % NUM_THREADS) nthreads++;
    // nthreads = nthreads * NUM_THREADS;
    // nthreads = biggest;
    
    int     *d_csrRowPtr;
    int     *d_csrColIdx;
    double  *d_csrVal;
   
    int     *d_cscColPtr;
    int     *d_cscRowIdx;
    double  *d_cscVal;

    int     *d_csrRowIdx;    
    int     *d_intra;
    int     *d_inter;

    // int device;
    // hipGetDevice(&device);

    // struct hipDeviceProp_t props;
    // hipGetDeviceProperties(&props, device);

    // int maxThreads = props.multiProcessorCount * props.maxThreadsPerMultiProcessor;
    // int maxBlocks = props.multiProcessorCount * 16;    

    // int requiredBlocks = biggest / NUM_THREADS;
    // if(biggest % NUM_THREADS) requiredBlocks++;
    // int requiredThreads = requiredBlocks * NUM_THREADS;

    // int blockNum;

    // if (requiredThreads > maxThreads) {
    //     std::cout << "numero di thread richiesto eccessivo: " << requiredThreads << " su " << maxThreads << " " << std::endl;
    //     std::cout << "eseguiamo comunque il tutto limitando il numero di thread al massimo possibile ed eseguendo più chiamate ai kernel" << std::endl;
    //     std::cout << "questo potrebbe avere impatto sulle performance" << std::endl;
    //     blockNum = maxThreads / NUM_THREADS;
    //     std::cout << NUM_THREADS << " " << blockNum << std::endl;
    // }
    // else {
    //     blockNum = requiredBlocks;
    //     std::cout << blockNum << " " << biggest <<std::endl;
    // }

    unsigned long long int reqMem = (nnz * sizeof(int)) * 4 + (nnz * sizeof(double)) * 2 + (biggest + 1) * sizeof(int);
    double nvidiaFreeMemory = getSizeOfNvidiaFreeMemory();
    unsigned long long int actualFreeMem = nvidiaFreeMemory - reqMem;
    std::cout << std::setprecision(0) << "reqMem: " << reqMem << " free memory: " << nvidiaFreeMemory << " actual: " << actualFreeMem << std::endl;
    unsigned long long int altro = (unsigned long long int)((unsigned long long int)(biggest + 1) * (unsigned long long int)n) * sizeof(int);    
    // if ( (actualFreeMem / n) < biggest) {
    //     std::cout << "Questa matrice non ci sta in memoria con un numero di righe pari a " << biggest << " per cui la allocheremo più piccola" << std::endl;
    //     inter_dim = (actualFreeMem / n) - 1;
    //     std::cout << inter_dim << std::endl;
    // }
    std::cout << std::setprecision(0) << "altro: " << altro << " free memory: " << actualFreeMem << std::endl;
    if ( actualFreeMem < altro) {        
        return -1;
    }

    // questi ci serono ovunque
    hipMalloc(&d_csrColIdx, nnz     * sizeof(int));    
    hipMemcpy(d_csrColIdx, csrColIdx, nnz     * sizeof(int),    hipMemcpyHostToDevice);
    hipMalloc(&d_cscColPtr, (n + 1) * sizeof(int));    
    
    // questi ci servono nel primo kernel 
    hipMalloc(&d_csrRowPtr, (m + 1) * sizeof(int));
    hipMemcpy(d_csrRowPtr, csrRowPtr, (m + 1) * sizeof(int),    hipMemcpyHostToDevice);
    hipMalloc(&d_csrRowIdx, nnz                * sizeof(int));
    hipMalloc(&d_intra,     nnz                * sizeof(int));
    hipMalloc(&d_inter,     (inter_dim + 1) * n * sizeof(int));
    hipMemset(d_inter,     0, (inter_dim + 1) * n   * sizeof(int));

    int blockSize1;
    int minGridSize1;
    int gridSize1;
    int blockSize2;
    int minGridSize2;
    int gridSize2;
    int blockSize3;
    int minGridSize3;
    int gridSize3;

    hipOccupancyMaxPotentialBlockSize(&minGridSize1, &blockSize1, transpostionSpeedKernel, 0, biggest);
    hipOccupancyMaxPotentialBlockSize(&minGridSize2, &blockSize2, transpostionSpeedKernel1, 0, biggest);
    hipOccupancyMaxPotentialBlockSize(&minGridSize3, &blockSize3, transpostionSpeedKernel3, 0, biggest);

    gridSize1 = (biggest + blockSize1 - 1) / blockSize1;
    gridSize2 = (biggest + blockSize2 - 1) / blockSize2;
    gridSize3 = (biggest + blockSize3 - 1) / blockSize3;


    std::cout << "blockSize1: " << blockSize1 << " minGridSize1: " << minGridSize1  << " gridSize1: " << gridSize1 << std::endl;
    std::cout << "blockSize2: " << blockSize2 << " minGridSize2: " << minGridSize2  << " gridSize2: " << gridSize2 << std::endl;
    std::cout << "blockSize3: " << blockSize3 << " minGridSize3: " << minGridSize3  << " gridSize3: " << gridSize3 << std::endl;

    TM_device.start();

    // dim3 DimGrid(blockNum, 1, 1);
    // dim3 DimBlock(NUM_THREADS, 1, 1);

    transpostionSpeedKernel<<<gridSize1, blockSize1>>>(m, n, nnz, d_csrRowPtr, d_csrColIdx, d_cscColPtr, d_csrRowIdx, d_intra, d_inter, inter_dim);

    hipDeviceSynchronize();
    CHECK_CUDA_ERROR

    // non ci servirà più
    hipFree(d_csrRowPtr);

    transpostionSpeedKernel1<<<gridSize2, blockSize2>>>(m, n, nnz, d_csrColIdx, d_cscColPtr, d_csrRowIdx, d_intra, d_inter);

    hipDeviceSynchronize();

    std::cout << "### Before: " << getSizeOfNvidiaFreeMemory() << std::endl;
    
    int clean = manageMemoryForScan(n + 1);
    int *intra;
    int *csrRowIdx;
    if(clean == 0) {
        intra  = (int *)malloc(nnz * sizeof(int));
        hipMemcpy(intra, d_intra, (nnz) * sizeof(int),  hipMemcpyDeviceToHost);
        hipFree(d_intra);
        
        hipMemcpy(csrColIdx, d_csrColIdx, nnz * sizeof(int),    hipMemcpyDeviceToHost);
        hipFree(d_csrColIdx);
        
        csrRowIdx  = (int *)malloc(nnz * sizeof(int));
        hipMemcpy(csrRowIdx, d_csrRowIdx, nnz * sizeof(int),    hipMemcpyDeviceToHost);
        hipFree(d_csrRowIdx);
    }
    
    std::cout << "### After: " << getSizeOfNvidiaFreeMemory() << std::endl;
    
    fullPrescan(d_inter + (n * biggest), d_cscColPtr, n + 1);
    
    if(clean == 0) {
        hipMalloc(&d_intra,     nnz                * sizeof(int));
        hipMemcpy(d_intra, intra, nnz     * sizeof(int),    hipMemcpyHostToDevice);
    
        hipMalloc(&d_csrColIdx, nnz     * sizeof(int));    
        hipMemcpy(d_csrColIdx, csrColIdx, nnz     * sizeof(int),    hipMemcpyHostToDevice);
    
        hipMalloc(&d_csrRowIdx, nnz                * sizeof(int));
        hipMemcpy(d_csrRowIdx, csrRowIdx, nnz     * sizeof(int),    hipMemcpyHostToDevice);

        free(intra);
        free(csrRowIdx);
    }

    // questi ci servono nell'ultimo
    hipMalloc(&d_csrVal,    nnz     * sizeof(double));
    hipMemcpy(d_csrVal,    csrVal,    nnz     * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc(&d_cscRowIdx, nnz     * sizeof(int));
    hipMalloc(&d_cscVal,    nnz     * sizeof(double)); 

    hipDeviceSynchronize();   
    
    transpostionSpeedKernel3<<<gridSize3, blockSize3>>>(m, n, nnz, d_csrColIdx, d_csrVal, d_cscColPtr, d_cscRowIdx, d_cscVal, d_csrRowIdx, d_intra, d_inter);

    TM_device.stop();
    hipDeviceSynchronize();
    TM_device.print("GPU Sparse Matrix Transpostion ScanSpeed: ");    
  
    hipMemcpy(cscColPtr, d_cscColPtr, (n+1) * sizeof(int),  hipMemcpyDeviceToHost);
    hipMemcpy(cscRowIdx, d_cscRowIdx, nnz * sizeof(int),    hipMemcpyDeviceToHost);
    hipMemcpy(cscVal,    d_cscVal,    nnz * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_csrColIdx);
    hipFree(d_csrVal);

    hipFree(d_cscColPtr);
    hipFree(d_cscRowIdx);
    hipFree(d_cscVal);

    hipFree(d_csrRowIdx);
    hipFree(d_intra);
    hipFree(d_inter);


      ////////////////////////////////////// For debug ///////////////////////////////////////

    //   //int *intra  = (int *)malloc(nnz * sizeof(int));
    //   int *inter  = (int *)malloc((biggest + 1) * n * sizeof(int) * sizeof(int));
    //   //int *csrRowIdx  = (int *)malloc(nnz * sizeof(int));
    //   hipMemcpy(intra, d_intra, nnz * sizeof(int), hipMemcpyDeviceToHost);
    //   hipMemcpy(inter, d_inter, (biggest + 1) * n * sizeof(int), hipMemcpyDeviceToHost);
    //   hipMemcpy(csrRowIdx, d_csrRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost);
  
    //   std::cout << "intra: ";
    //   for(int i = 0; i < nnz; i++) {
    //       std::cout << intra[i] << " ";
    //   }
    //   std::cout << std::endl;
  
    //   std::cout << "inter: ";
    //   for(int i = 0; i < ((biggest + 1) * n); i++) {
    //       std::cout << inter[i] << " ";
    //   }
    //   std::cout << std::endl;
      
    //   std::cout << "csrRowIdx: ";
    //   for(int i = 0; i < nnz; i++) {
    //       std::cout << csrRowIdx[i] << " ";
    //   }
    //   std::cout << std::endl;
  
    //   //free(intra);
    //   free(inter);
    //   //free(csrRowIdx);
  
      ////////////////////////////////////////////////////////////////////////////////////////




    return TM_device.duration(); 
}
