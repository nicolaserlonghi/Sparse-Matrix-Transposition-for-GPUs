#include <nvidia.h>
#include <utilities.h>
#include <Timer.cuh>

using namespace timer;

float nvidia(
    int m,
    int n,
    int nnz,
    int *csrRowPtr,
    int *csrColIdx,
    double *csrVal,
    int *cscColPtr,
    int *cscRowIdx,
    double *cscVal
) {
    hipSetDevice(0);
    // Check if memory is enough
    double reqMem = (nnz * sizeof(int)) * 2 + (nnz * sizeof(double)) * 2 + (m+1) * sizeof(int) + (n+1) * sizeof(int);
    double nvidiaFreeMemory = getSizeOfNvidiaFreeMemory();    
    if ( nvidiaFreeMemory < reqMem)
        return -1;
        
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);

    int     *d_csrRowPtr;
    int     *d_csrColIdx;
    double  *d_csrVal;
    int     *d_cscColPtr;
    int     *d_cscRowIdx;
    double  *d_cscVal;
    // Set host memory
    hipMalloc(&d_cscColPtr, (n+1) * sizeof(int));
    hipMalloc(&d_cscRowIdx, nnz   * sizeof(int));
    hipMalloc(&d_cscVal,    nnz   * sizeof(double));
    hipMalloc(&d_csrRowPtr, (m+1) * sizeof(int));
    hipMalloc(&d_csrColIdx, nnz   * sizeof(int));
    hipMalloc(&d_csrVal,    nnz   * sizeof(double));
    hipMemcpy(d_csrRowPtr, csrRowPtr, (m+1) * sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIdx, csrColIdx, nnz   * sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal,    csrVal,    nnz   * sizeof(double), hipMemcpyHostToDevice);
    
    // setup buffersize
    size_t  P_bufferSize = 0;
    char*   p_buffer= NULL;

    hipsparseCsr2cscEx2_bufferSize(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1,
        &P_bufferSize
    );

    reqMem = reqMem + static_cast<double>(P_bufferSize);
    if (nvidiaFreeMemory < reqMem) {
        hipFree(d_csrRowPtr);
        hipFree(d_csrColIdx);
        hipFree(d_csrVal);
        hipFree(d_cscColPtr);
        hipFree(d_cscRowIdx);
        hipFree(d_cscVal);
        return -1;
    }
    hipMalloc(&p_buffer, P_bufferSize);

    // Start computation
    Timer<DEVICE> TM_device;
    TM_device.start();
    hipsparseCsr2cscEx2(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1,
        p_buffer
    );
    // Take time
    TM_device.stop();
    TM_device.print("GPU Sparse Matrix Transpostion ALGO1: ");
    // Get result from host
    hipMemcpy(cscColPtr, d_cscColPtr, (n+1) * sizeof(int),  hipMemcpyDeviceToHost);
    hipMemcpy(cscRowIdx, d_cscRowIdx, nnz * sizeof(int),    hipMemcpyDeviceToHost);
    hipMemcpy(cscVal,    d_cscVal,    nnz * sizeof(double), hipMemcpyDeviceToHost);
    // Cleaner
    hipsparseDestroy(handle);
    hipFree(d_csrRowPtr);
    hipFree(d_csrColIdx);
    hipFree(d_csrVal);
    hipFree(d_cscColPtr);
    hipFree(d_cscRowIdx);
    hipFree(d_cscVal);

    return TM_device.duration(); 
}


float nvidia2(
    int     m,
    int     n,
    int     nnz,
    int     *csrRowPtr,
    int     *csrColIdx,
    double  *csrVal,
    int     *cscColPtr,
    int     *cscRowIdx,
    double  *cscVal
) {
    hipSetDevice(0);
    // Check if memory is enough
    double reqMem = (nnz * sizeof(int)) * 2 + (nnz * sizeof(double)) * 2 + (m+1) * sizeof(int) + (n+1) * sizeof(int);
    double nvidiaFreeMemory = getSizeOfNvidiaFreeMemory();    
    if ( nvidiaFreeMemory < reqMem) {
        return -1;
    }
        
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);

    int     *d_csrRowPtr;
    int     *d_csrColIdx;
    double  *d_csrVal;
    int     *d_cscColPtr;
    int     *d_cscRowIdx;
    double  *d_cscVal;
    // Set host memory
    hipMalloc(&d_cscColPtr, (n+1) * sizeof(int));
    hipMalloc(&d_cscRowIdx, nnz   * sizeof(int));
    hipMalloc(&d_cscVal,    nnz   * sizeof(double));
    hipMalloc(&d_csrRowPtr, (m+1) * sizeof(int));
    hipMalloc(&d_csrColIdx, nnz   * sizeof(int));
    hipMalloc(&d_csrVal,    nnz   * sizeof(double));
    hipMemcpy(d_csrRowPtr, csrRowPtr, (m+1) * sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIdx, csrColIdx, nnz   * sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal,    csrVal,    nnz   * sizeof(double), hipMemcpyHostToDevice);
    
    // setup buffersize
    size_t  P_bufferSize = 0;
    char*   p_buffer= NULL;

    hipsparseCsr2cscEx2_bufferSize(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG2,
        &P_bufferSize
    );

    reqMem = reqMem + static_cast<double>(P_bufferSize);
    if (nvidiaFreeMemory < reqMem) {
        hipFree(d_csrRowPtr);
        hipFree(d_csrColIdx);
        hipFree(d_csrVal);
        hipFree(d_cscColPtr);
        hipFree(d_cscRowIdx);
        hipFree(d_cscVal);
        return -1;
    }
    hipMalloc(&p_buffer, P_bufferSize);
    
    // Start computation
    Timer<DEVICE> TM_device;
    TM_device.start();
    hipsparseCsr2cscEx2(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG2,
        p_buffer
    );
    // Take time
    TM_device.stop();
    TM_device.print("GPU Sparse Matrix Transpostion ALGO2: ");
    // Copy result from host
    hipMemcpy(cscColPtr, d_cscColPtr, (n+1) * sizeof(int),  hipMemcpyDeviceToHost);
    hipMemcpy(cscRowIdx, d_cscRowIdx, nnz * sizeof(int),    hipMemcpyDeviceToHost);
    hipMemcpy(cscVal,    d_cscVal,    nnz * sizeof(double), hipMemcpyDeviceToHost);
    // Cleaner
    hipsparseDestroy(handle);
    hipFree(d_csrRowPtr);
    hipFree(d_csrColIdx);
    hipFree(d_csrVal);
    hipFree(d_cscColPtr);
    hipFree(d_cscRowIdx);
    hipFree(d_cscVal);

    return TM_device.duration();
}