#include <nvidia.h>
#include <utilities.h>
#include <Timer.cuh>
#include <CheckError.cuh>

using namespace timer;

using namespace std;


void cuda_sptrans(
    int         m,
    int         n,
    int         nnz,
    int        *csrRowPtr,
    int        *csrColIdx,
    double     *csrVal,
    int        *cscColPtr,
    int        *cscRowIdx,
    double     *cscVal
) {
    Timer<DEVICE> TM_device;
    hipSetDevice(0);
    
    hipsparseHandle_t handle = NULL;

    hipsparseCreate(&handle);

    int *d_csrRowPtr;
    int *d_csrColIdx;
    double *d_csrVal;
   
    int *d_cscColPtr;
    int *d_cscRowIdx;
    double *d_cscVal;

    // Qui inizia il calcolo del tempo di copia dei dati

    // TODO: verificare cosa fa il mallocManaged rispetto a Malloc
    // TODO: verificare perché serve proprio memset

    // Matrix csr
    SAFE_CALL( hipMallocManaged((void **)&d_csrRowPtr, (m+1) * sizeof(int)) );
    SAFE_CALL( hipMallocManaged((void **)&d_csrColIdx, nnz   * sizeof(int)) );
    SAFE_CALL( hipMallocManaged((void **)&d_csrVal,    nnz   * sizeof(double)) );

    SAFE_CALL( hipMemcpy(d_csrRowPtr, csrRowPtr, (m+1) * sizeof(int),   hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(d_csrColIdx, csrColIdx, nnz  * sizeof(int),    hipMemcpyHostToDevice) );
    SAFE_CALL( hipMemcpy(d_csrVal,    csrVal,    nnz  * sizeof(double), hipMemcpyHostToDevice) );

    // Matrix csc     
    SAFE_CALL( hipMallocManaged((void **)&d_cscColPtr, (n+1) * sizeof(int)) );
    SAFE_CALL( hipMallocManaged((void **)&d_cscRowIdx, nnz   * sizeof(int)) );
    SAFE_CALL( hipMallocManaged((void **)&d_cscVal,    nnz   * sizeof(double)) );

    SAFE_CALL( hipMemset(d_cscColPtr, 0, (n+1) * sizeof(int)) );
    SAFE_CALL( hipMemset(d_cscRowIdx, 0, nnz   * sizeof(int)) );
    SAFE_CALL( hipMemset(d_cscVal,    0, nnz   * sizeof(double)) );

    // Qui finisce il tempo per la copia dei dati
    
    // setup buffersize

    TM_device.start();

    // Qui andrebbero i DimGrid e DimBlock

    size_t P_bufferSize = 0;

    char* p_buffer= NULL;

    hipsparseCsr2cscEx2_bufferSize(
                            handle,
                            m,
                            n,
                            nnz,
                            d_csrVal,
                            d_csrRowPtr,
                            d_csrColIdx,
                            d_cscVal,
                            d_cscColPtr,
                            d_cscRowIdx,
                            HIP_C_32F,
                            HIPSPARSE_ACTION_NUMERIC,
                            HIPSPARSE_INDEX_BASE_ZERO,
                            HIPSPARSE_CSR2CSC_ALG1,
                            &P_bufferSize);

    printf("P_bufferSize  = %lld \n", (long long)P_bufferSize);

    if (NULL != p_buffer) { 
        SAFE_CALL( hipFree(p_buffer) );
    }

    SAFE_CALL( hipMalloc((void**)&p_buffer, P_bufferSize) );
    
    hipsparseCsr2cscEx2(
                handle,
                m,
                n,
                nnz,
                d_csrVal,
                d_csrRowPtr,
                d_csrColIdx,
                d_cscVal,
                d_cscColPtr,
                d_cscRowIdx,
		        HIP_C_32F,
                HIPSPARSE_ACTION_NUMERIC,
                HIPSPARSE_INDEX_BASE_ZERO,
                HIPSPARSE_CSR2CSC_ALG1,
                p_buffer);

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("GPU Sparse Matrix Transpostion: ");

  
    SAFE_CALL( hipMemcpy(cscColPtr, d_cscColPtr, (n+1) * sizeof(int),  hipMemcpyDeviceToHost) );
    SAFE_CALL( hipMemcpy(cscRowIdx, d_cscRowIdx, nnz * sizeof(int),    hipMemcpyDeviceToHost) );
    SAFE_CALL( hipMemcpy(cscVal,    d_cscVal,    nnz * sizeof(double), hipMemcpyDeviceToHost) );

    // step 6: free resources

    hipsparseDestroy(handle);
    SAFE_CALL( hipFree(d_csrRowPtr) );
    SAFE_CALL( hipFree(d_csrColIdx) );
    SAFE_CALL( hipFree(d_csrVal) );
    SAFE_CALL( hipFree(d_cscColPtr) );
    SAFE_CALL( hipFree(d_cscRowIdx) );
    SAFE_CALL( hipFree(d_cscVal) );

}