#include <nvidia.h>
#include <utilities.h>
#include <Timer.cuh>

using namespace timer;

float nvidia(
    int m,
    int n,
    int nnz,
    int *csrRowPtr,
    int *csrColIdx,
    double *csrVal,
    int *cscColPtr,
    int *cscRowIdx,
    double *cscVal
) {
    hipError_t err = hipSuccess;
    hipSetDevice(0);
    // Check if memory is enough
    double reqMem = (nnz * sizeof(int)) * 2 + (nnz * sizeof(double)) * 2 + (m+1) * sizeof(int) + (n+1) * sizeof(int);
    double nvidiaFreeMemory = getSizeOfNvidiaFreeMemory();    
    if ( nvidiaFreeMemory < reqMem)
        return -1;
        
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);

    int     *d_csrRowPtr;
    int     *d_csrColIdx;
    double  *d_csrVal;
    int     *d_cscColPtr;
    int     *d_cscRowIdx;
    double  *d_cscVal;
    // Set device memory
    err = hipMalloc(&d_cscColPtr, (n+1) * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_cscColPtr");
    err = hipMalloc(&d_cscRowIdx, nnz * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_cscRowIdx");
    err = hipMalloc(&d_cscVal, nnz * sizeof(double));
    CUDA_ERROR(err, "Failed to allocate device vector d_cscVal");
    err = hipMalloc(&d_csrRowPtr, (m+1) * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_csrRowPtr");
    err = hipMalloc(&d_csrColIdx, nnz * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_csrColIdx");
    err = hipMalloc(&d_csrVal, nnz * sizeof(double));
    CUDA_ERROR(err, "Failed to allocate device vector d_csrVal");
    err = hipMemcpy(d_csrRowPtr, csrRowPtr, (m+1) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERROR(err, "Failed to copy array csrRowPtr from host to device");
    err = hipMemcpy(d_csrColIdx, csrColIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERROR(err, "Failed to copy array csrColIdx from host to device");
    err = hipMemcpy(d_csrVal, csrVal, nnz * sizeof(double), hipMemcpyHostToDevice);
    CUDA_ERROR(err, "Failed to copy array csrVal from host to device");
    
    // setup buffersize
    size_t  P_bufferSize = 0;
    char*   p_buffer= NULL;

    hipsparseCsr2cscEx2_bufferSize(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1,
        &P_bufferSize
    );

    reqMem = reqMem + static_cast<double>(P_bufferSize);
    if (nvidiaFreeMemory < reqMem) {
        err = hipFree(d_csrRowPtr);
        CUDA_ERROR(err, "Failed to free device array d_csrRowPtr");
        err = hipFree(d_csrColIdx);
        CUDA_ERROR(err, "Failed to free device array d_csrColIdx");
        err = hipFree(d_csrVal);
        CUDA_ERROR(err, "Failed to free device array d_csrVal");
        err = hipFree(d_cscColPtr);
        CUDA_ERROR(err, "Failed to free device array d_cscColPtr");
        err = hipFree(d_cscRowIdx);
        CUDA_ERROR(err, "Failed to free device array d_cscRowIdx");
        err = hipFree(d_cscVal);
        CUDA_ERROR(err, "Failed to free device array d_cscVal");
        return -1;
    }
    err = hipMalloc(&p_buffer, P_bufferSize);
    CUDA_ERROR(err, "Failed to allocate device vector p_buffer");

    // Start computation
    Timer<DEVICE> TM_device;
    TM_device.start();
    hipsparseCsr2cscEx2(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1,
        p_buffer
    );
    err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch hipsparseCsr2cscEx2 algo 1");
    // Take time
    TM_device.stop();
    TM_device.print("GPU Sparse Matrix Transpostion ALGO1: ");
    // Get result from device
    err = hipMemcpy(cscColPtr, d_cscColPtr, (n+1) * sizeof(int), hipMemcpyDeviceToHost);
    CUDA_ERROR(err, "Failed to copy array d_cscColPtr from device to host");
    err = hipMemcpy(cscRowIdx, d_cscRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost);
    CUDA_ERROR(err, "Failed to copy array d_cscRowIdx from device to host");
    err = hipMemcpy(cscVal, d_cscVal, nnz * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_ERROR(err, "Failed to copy array d_cscVal from device to host");
    // Cleaner
    hipsparseDestroy(handle);
    err = hipFree(d_csrRowPtr);
    CUDA_ERROR(err, "Failed to free device array d_csrRowPtr");
    err = hipFree(d_csrColIdx);
    CUDA_ERROR(err, "Failed to free device array d_csrColIdx");
    err = hipFree(d_csrVal);
    CUDA_ERROR(err, "Failed to free device array d_csrVal");
    err = hipFree(d_cscColPtr);
    CUDA_ERROR(err, "Failed to free device array d_cscColPtr");
    err = hipFree(d_cscRowIdx);
    CUDA_ERROR(err, "Failed to free device array d_cscRowIdx");
    err = hipFree(d_cscVal);
    CUDA_ERROR(err, "Failed to free device array d_cscVal");

    return TM_device.duration(); 
}


float nvidia2(
    int     m,
    int     n,
    int     nnz,
    int     *csrRowPtr,
    int     *csrColIdx,
    double  *csrVal,
    int     *cscColPtr,
    int     *cscRowIdx,
    double  *cscVal
) {
    hipSetDevice(0);
    // Check if memory is enough
    double reqMem = (nnz * sizeof(int)) * 2 + (nnz * sizeof(double)) * 2 + (m+1) * sizeof(int) + (n+1) * sizeof(int);
    double nvidiaFreeMemory = getSizeOfNvidiaFreeMemory();    
    if ( nvidiaFreeMemory < reqMem) {
        return -1;
    }
        
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);

    int     *d_csrRowPtr;
    int     *d_csrColIdx;
    double  *d_csrVal;
    int     *d_cscColPtr;
    int     *d_cscRowIdx;
    double  *d_cscVal;
    // Set device memory
    err = hipMalloc(&d_cscColPtr, (n+1) * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_cscColPtr");
    err = hipMalloc(&d_cscRowIdx, nnz   * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_cscRowIdx");
    err = hipMalloc(&d_cscVal,    nnz   * sizeof(double));
    CUDA_ERROR(err, "Failed to allocate device vector d_cscVal");
    err = hipMalloc(&d_csrRowPtr, (m+1) * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_csrRowPtr");
    err = hipMalloc(&d_csrColIdx, nnz   * sizeof(int));
    CUDA_ERROR(err, "Failed to allocate device vector d_csrColIdx");
    err = hipMalloc(&d_csrVal,    nnz   * sizeof(double));
    CUDA_ERROR(err, "Failed to allocate device vector d_csrVal");
    err = hipMemcpy(d_csrRowPtr, csrRowPtr, (m+1) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERROR(err, "Failed to copy array csrRowPtr from host to device");
    err = hipMemcpy(d_csrColIdx, csrColIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERROR(err, "Failed to copy array csrColIdx from host to device");
    err = hipMemcpy(d_csrVal, csrVal, nnz * sizeof(double), hipMemcpyHostToDevice);
    CUDA_ERROR(err, "Failed to copy array csrVal from host to device");
    
    // setup buffersize
    size_t  P_bufferSize = 0;
    char*   p_buffer= NULL;

    hipsparseCsr2cscEx2_bufferSize(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG2,
        &P_bufferSize
    );

    reqMem = reqMem + static_cast<double>(P_bufferSize);
    if (nvidiaFreeMemory < reqMem) {
        err = hipFree(d_csrRowPtr);
        CUDA_ERROR(err, "Failed to free device array d_csrRowPtr");
        err = hipFree(d_csrColIdx);
        CUDA_ERROR(err, "Failed to free device array d_csrColIdx");
        err = hipFree(d_csrVal);
        CUDA_ERROR(err, "Failed to free device array d_csrVal");
        err = hipFree(d_cscColPtr);
        CUDA_ERROR(err, "Failed to free device array d_cscColPtr");
        err = hipFree(d_cscRowIdx);
        CUDA_ERROR(err, "Failed to free device array d_cscRowIdx");
        err = hipFree(d_cscVal);
        CUDA_ERROR(err, "Failed to free device array d_cscVal");
        return -1;
    }
    err = hipMalloc(&p_buffer, P_bufferSize);
    CUDA_ERROR(err, "Failed to allocate device vector p_buffer");
    
    // Start computation
    Timer<DEVICE> TM_device;
    TM_device.start();
    hipsparseCsr2cscEx2(
        handle,
        m,
        n,
        nnz,
        d_csrVal,
        d_csrRowPtr,
        d_csrColIdx,
        d_cscVal,
        d_cscColPtr,
        d_cscRowIdx,
        HIP_R_64F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG2,
        p_buffer
    );
    err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch hipsparseCsr2cscEx2 algo 2");
    // Take time
    TM_device.stop();
    TM_device.print("GPU Sparse Matrix Transpostion ALGO2: ");
    // Copy result from device
    err = hipMemcpy(cscColPtr, d_cscColPtr, (n+1) * sizeof(int), hipMemcpyDeviceToHost);
    CUDA_ERROR(err, "Failed to copy array d_cscColPtr from device to host");
    err = hipMemcpy(cscRowIdx, d_cscRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost);
    CUDA_ERROR(err, "Failed to copy array d_cscRowIdx from device to host");
    err = hipMemcpy(cscVal, d_cscVal, nnz * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_ERROR(err, "Failed to copy array d_cscVal from device to host");
    // Cleaner
    hipsparseDestroy(handle);
    err = hipFree(d_csrRowPtr);
    CUDA_ERROR(err, "Failed to free device array d_csrRowPtr");
    err = hipFree(d_csrColIdx);
    CUDA_ERROR(err, "Failed to free device array d_csrColIdx");
    err = hipFree(d_csrVal);
    CUDA_ERROR(err, "Failed to free device array d_csrVal");
    err = hipFree(d_cscColPtr);
    CUDA_ERROR(err, "Failed to free device array d_cscColPtr");
    err = hipFree(d_cscRowIdx);
    CUDA_ERROR(err, "Failed to free device array d_cscRowIdx");
    err = hipFree(d_cscVal);
    CUDA_ERROR(err, "Failed to free device array d_cscVal");

    return TM_device.duration();
}